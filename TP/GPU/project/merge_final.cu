
#include <hip/hip_runtime.h>
#include <ctime>
#include <stdio.h>
#include <math.h>
#include <limits.h>

#define NB 1
#define NTPB 1024

#define lenC 50


// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line){
	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value 
// of the macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))


__global__ void merge(int *a, int *b, int *c, int d, int len_new){
	// A represents the biggest array

    __shared__ int A_copy[lenC];
    __shared__ int B_copy[lenC];
    __shared__ int C_copy[lenC];

    for(int i=0;i<lenC;i++){
        C_copy[i] = c[i];
    }

	int *A, *B, *C, la, lb, offset;
    int Kx, Ky, Px, Py, Qx, Qy;

    A = a;
    B = b;
    C = c;

    int block = 0;
    int i = threadIdx.x;
    int i_d = i%d;
    int block_d = i/d;

    int start_x, start_y, end_x, end_y;

    if(i>=len_new)
        return;

    start_x = d*block_d;
    start_y = d*(block_d+1);

    end_x = d*(block_d+1);
    end_y = d*(block_d+2);

    if(block_d%2 == 0)
        A[i] = c[i];
    else
        B[i-d] = c[i];
}

void wrapper_partition(int *A, int *B, int *C, int p, int len_new){
    int *aGPU, *bGPU, *cGPU;
    float TimerV;
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
    testCUDA(hipEventCreate(&stop));
    testCUDA(hipEventRecord(start, 0));

    testCUDA(hipMalloc(&aGPU, len_new*sizeof(int)));
    testCUDA(hipMalloc(&bGPU, len_new*sizeof(int)));
    testCUDA(hipMalloc(&cGPU, len_new*sizeof(int)));   
    testCUDA(hipMemcpy(cGPU, C, len_new*sizeof(int), hipMemcpyHostToDevice));

    //start of kernel
    // partition<<<NB, 1>>>(aGPU, bGPU, a_partGPU, b_partGPU, len_A, len_B);
    
    // for(int l = 0; l<=8; l++)
    merge<<<1, NTPB>>>(aGPU, bGPU, cGPU, 8, len_new);

    testCUDA(hipMemcpy(C, cGPU, len_new*sizeof(int), hipMemcpyDeviceToHost));
    testCUDA(hipMemcpy(A, aGPU, len_new*sizeof(int), hipMemcpyDeviceToHost));
    testCUDA(hipMemcpy(B, bGPU, len_new*sizeof(int), hipMemcpyDeviceToHost));

    testCUDA(hipEventRecord(stop, 0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimerV, start, stop));

    printf("Execution time: %f ms\n", TimerV);

    testCUDA(hipFree(aGPU));
    testCUDA(hipFree(bGPU));
    testCUDA(hipFree(cGPU));
    testCUDA(hipEventDestroy(start));
    testCUDA(hipEventDestroy(stop));
}

int main(void){
    // On veut trier C!!
    // on va copier sur A et B des morceaux de C 
    int p = (int) floor(log2(lenC));
    int to_add = ((int) pow(2, p+1))%lenC;

    int len_new = lenC+ to_add;

    int *A = (int*)malloc(len_new* sizeof(int));
    int *B = (int*)malloc(len_new* sizeof(int));
    int *C = (int*)malloc(len_new* sizeof(int));

    for(int i=0; i<lenC; i++){
        C[i] = rand() %250;
	}
    for(int i=lenC; i<lenC + to_add; i++){
        C[i] = INT_MAX;
	}

    wrapper_partition(A, B, C, p, len_new);

    printf("\n");
    for(int i = 0; i < len_new ; i++){
        printf("| %d |", C[i], i);
    }
    printf("\n\n");
    for(int i = 0; i < len_new ; i++){
        printf("| %d |", A[i], i);
    }
    printf("\n\n");
    for(int i = 0; i < len_new ; i++){
        printf("| %d |", B[i], i);
    }

    return 0;
}