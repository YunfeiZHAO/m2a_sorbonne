 #include <ctime>
#include <stdio.h>
#include <math.h>
#include <limits.h>
#include <hip/hip_runtime.h>

#define NTPB 1000
#define lenC 2000

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line){
	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value 
// of the macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))


__global__ void merge(int *C, int d, int len_new){

    __shared__ int C_sh[NTPB];

	int la, lb, offset;
    int Kx, Ky, Px, Py, Qx, Qy;

    int i = threadIdx.x;
    int tidx = threadIdx.x%d;
    int Qt = (threadIdx.x -tidx)/d;
    int gbx = Qt + blockIdx.x*(blockDim.x/d);
    
    if((gbx*d + tidx)>=len_new)
        return;

    C_sh[Qt*d + tidx] = C[gbx*d + tidx];
    __syncthreads();

	la = d/2;
	lb = d/2;
    
    if(tidx > la){
		Kx= tidx-la;
		Ky= la;
		Px= la;
		Py= tidx-la;
	}
	else{
		Kx= 0;
		Ky= tidx;
		Px= tidx;
		Py= 0;
    }

    int move_a = d*Qt;
    int move_b = d*Qt + d/2;

	while(true){
		offset = (Ky-Py)/2;
		Qx = Kx + offset;
		Qy = Ky - offset;
		if( Qx<= lb && (  Qy==la || Qx==0  || C_sh[Qy+move_a]>C_sh[Qx-1+move_b])){
			if(Qx == lb || Qy == 0 || C_sh[Qy-1+ move_a]<=C_sh[Qx+ move_b] ){
				if(Qy< la && (Qx == lb || C_sh[Qy+move_a] <= C_sh[Qx+move_b])){
          C[gbx*d + tidx]= C_sh[Qy+move_a];
        }else{
          C[gbx*d + tidx]= C_sh[Qx+move_b]; 
        }
				break;
			}else{
				Kx= Qx+1;
				Ky = Qy-1;		
			}
		}else{
			Px= Qx-1;
			Py = Qy+1;		
		}
	}  
}

 void wrapper_partition(int *C, int NB,int len_new, int p){
    int *cGPU;
    float TimerV;
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
    testCUDA(hipEventCreate(&stop));
    testCUDA(hipEventRecord(start, 0));

    testCUDA(hipMalloc(&cGPU, len_new*sizeof(int)));   
    testCUDA(hipMemcpy(cGPU, C, len_new*sizeof(int), hipMemcpyHostToDevice));

    for(int l = 1; l<= p; l++){ 
        merge<<<NB, pow(2,p) >>>(cGPU, pow(2,l),len_new);
    }

    testCUDA(hipMemcpy(C, cGPU, len_new*sizeof(int), hipMemcpyDeviceToHost));

    testCUDA(hipEventRecord(stop, 0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimerV, start, stop));

    printf("Execution time: %f ms\n", TimerV);

    testCUDA(hipFree(cGPU));
    testCUDA(hipEventDestroy(start));
    testCUDA(hipEventDestroy(stop));
}

int main(void){

	int p = (int) floor(log2(NTPB));
    int NB = (int) ceil(lenC/(float)pow(2,p));
	int len_new = (int) pow(2, (int) ceil(log2(lenC)));

    printf("NB: %d\n", NB);

    int *C = (int*) malloc(lenC* sizeof(int));
     for(int i=0; i<lenC; i++){
        C[i] = 3000-i;
	}

    for(int i=lenC; i<len_new; i++){
        C[i] = INT_MAX;
	} 
    
    wrapper_partition(C, NB, len_new,p);

    printf("\n");
    for(int i = 0; i < len_new ; i++){
        printf("| %d ", C[i]);
    }
    printf("\n");

    return 0;
}