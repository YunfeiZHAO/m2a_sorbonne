#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.h"


void addVect(int *a, int *b, int *c, int length){

	int i;

	for(i=0; i<length; i++){
		c[i] = a[i] + b[i];
	}
}


__global__ void addVect_k(int *a, int *b, int *c, int length){

	int i = threadIdx.x + blockIdx.x*blockDim.x;

	/*while(i<length){
		c[i] = a[i] + b[i];
		i += blockDim.x*gridDim.x;
    }*/

	if(i<length){
		c[i] = a[i] + b[i];
    }
}



int main (void){

	// Variables definition
	int *a, *b, *c;
	int *aGPU, *bGPU, *cGPU;
	int i;
	
	// Length for the size of arrays
	int length = 1e8;

	Timer Tim;							// CPU timer instructions

	// Memory allocation of arrays 
	a = (int*)malloc(length*sizeof(int));
	b = (int*)malloc(length*sizeof(int));
	c = (int*)malloc(length*sizeof(int));

	hipMalloc(&aGPU, length*sizeof(int));
	hipMalloc(&bGPU, length*sizeof(int));
	hipMalloc(&cGPU, length*sizeof(int));

	// Setting values
	for(i=0; i<length; i++){
		a[i] = i;
		b[i] = 9*i;
	}

	hipMemcpy(aGPU, a, length*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(bGPU, b, length*sizeof(int), hipMemcpyHostToDevice);


	Tim.start();						// CPU timer instructions

	// Executing the addition 
	//addVect(a, b, c, length);

	float TimeVar;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	//addVect_k<<<64, 256>>>(aGPU, bGPU, cGPU, length);


	addVect_k<<<(length+255)/256, 256>>>(aGPU, bGPU, cGPU, length);


	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&TimeVar, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);



	Tim.add();							// CPU timer instructions



	hipMemcpy(c, cGPU, length*sizeof(int), hipMemcpyDeviceToHost);

	// Displaying the results to check the correctness 
	for(i=length-50; i<length-45; i++){
		printf(" ( %i ): %i\n", a[i]+b[i], c[i]);
	}

	//printf("CPU Timer for the addition on the CPU of vectors: %f s\n", 
	//	   (float)Tim.getsum());			// CPU timer instructions

	printf("GPU Timer for the addition on the GPU of vectors: %f s\n", 
		   TimeVar/1000.0f);			// CPU timer instructions


	// Freeing the memory
	hipFree(aGPU);
	hipFree(bGPU);
	hipFree(cGPU);
	free(a);
	free(b);
	free(c);

	return 0;
}