#include "hip/hip_runtime.h"
/**************************************************************
Lokman A. Abbas-Turki code

Those who re-use this code should mention in their code 
the name of the author above.
***************************************************************/
#include "rng.h"


// Generate uniformly distributed random variables
__device__ void CMRG_d(int *a0, int *a1, int *a2, int *a3, int *a4, 
			     int *a5, float *g0, float *g1, int nb){

 const int m1 = 2147483647;// Requested for the simulation
 const int m2 = 2145483479;// Requested for the simulation
 int h, p12, p13, p21, p23, k, loc;// Requested local parameters

 for(k=0; k<nb; k++){
	 // First Component 
	 h = *a0/q13; 
	 p13 = a13*(h*q13-*a0)-h*r13;
	 h = *a1/q12; 
	 p12 = a12*(*a1-h*q12)-h*r12;

	 if (p13 < 0) {
	   p13 = p13 + m1;
	 }
	 if (p12 < 0) {
	   p12 = p12 + m1;
	 }
	 *a0 = *a1;
	 *a1 = *a2;
	 if( (p12 - p13) < 0){
	   *a2 = p12 - p13 + m1;  
	 } else {
	   *a2 = p12 - p13;
	 }
  
	 // Second Component 
	 h = *a3/q23; 
	 p23 = a23*(h*q23-*a3)-h*r23;
	 h = *a5/q21; 
	 p21 = a21*(*a5-h*q21)-h*r21;

	 if (p23 < 0){
	   p23 = p23 + m2;
	 }
	 if (p12 < 0){
	   p21 = p21 + m2;
	 }
	 *a3 = *a4;
	 *a4 = *a5;
	 if ( (p21 - p23) < 0) {
	   *a5 = p21 - p23 + m2;  
	 } else {
	   *a5 = p21 - p23;
	 }

	 // Combines the two MRGs
	 if(*a2 < *a5){
		loc = *a2 - *a5 + m1;
	 }else{loc = *a2 - *a5;} 

	 if(k){
		if(loc == 0){
			*g1 = Invmp*m1;
		}else{*g1 = Invmp*loc;}
	 }else{
		*g1 = 0.0f; 
		if(loc == 0){
			*g0 = Invmp*m1;
		}else{*g0 = Invmp*loc;}
	 }
  }
}

// Genrates Gaussian distribution from a uniform one (Box-Muller)
__device__ void BoxMuller_d(float *g0, float *g1){

  float loc;
  if (*g1 < 1.45e-6f){
    loc = sqrtf(-2.0f*logf(0.00001f))*cosf(*g0*2.0f*MoPI);
  } else {
    if (*g1 > 0.99999f){
      loc = 0.0f;
    } else {loc = sqrtf(-2.0f*logf(*g1))*cosf(*g0*2.0f*MoPI);}
  }
  *g0 = loc;
}

// Black & Scholes model
__device__ void BS_d(float *S2, float S1, float r,
					 float sigma, float dt, float e){

  *S2 = S1*expf((r-0.5f*sigma*sigma)*dt*dt + sigma*dt*e);
}

// Principal MC routine
__global__ void MC_k(float *S, float x_0, float r,
					 float sigma, float dt, int P1, int P2,
					 float K, float *R1, float *R2, float* U, 
					 int *It, float B, int Ntraj, int M,
					 TabSeedCMRG_t *pt_cmrg){

   int idx = threadIdx.x + blockIdx.x*blockDim.x;

   S[idx] = x_0;
   It[idx] = 0;
   for (int k=1; k<=M; k++){
	   CMRG_d(pt_cmrg[0][idx], pt_cmrg[0][idx]+1, pt_cmrg[0][idx]+2, 
			  pt_cmrg[0][idx]+3, pt_cmrg[0][idx]+4, pt_cmrg[0][idx]+5, 
			  U+idx, U+idx+Ntraj, 2);

	   BoxMuller_d(U+idx, U+idx+Ntraj);
	   BS_d(S+idx+(k%2)*Ntraj, S[idx+((k+1)%2)*Ntraj], r, sigma, dt, U[idx]);
	   /**************************************************************
		Step 4:
		-------
			Write the appropriate call of BoxMuller_d and BS_d 

	   ***************************************************************/
	   It[idx] += (S[idx+(k%2)*Ntraj]<B);
   }
   R1[idx] = expf(-r*dt*dt*M)*fmaxf(0.0f, S[idx+(M%2)*Ntraj]-K)*((It[idx]<=P2)&&(It[idx]>=P1));
   R2[idx] = R1[idx]*R1[idx];
}


int main()
{

	float T = 1.0f;
	float K = 100.0f;
	float x_0 = 100.0f;
	float vol = 0.2f;
	float r = 0.1f;
	float B = 120.0f;
	int M = 100;
	int P1 = 10;
	int P2 = 49;
	float dt = sqrtf(T/M);
	float sum = 0.0f;	
	float sum2 = 0.0f;
	float Tim;							// GPU timer instructions
	hipEvent_t start, stop;			// GPU timer instructions
	float *res1, *res2, *res1C, *res2C, *Rando, *Stock;
	int *It;
	int Ntraj = 512*512;

	hipMalloc(&res1, Ntraj*sizeof(float));
	hipMalloc(&res2, Ntraj*sizeof(float));
	res1C = (float*)malloc(Ntraj*sizeof(float));
	res2C = (float*)malloc(Ntraj*sizeof(float));

   /************************************************************
	Step 1:
	-------
		Allocate appropriately Rando, Stock and It

   *************************************************************/

	hipMalloc(&Rando, 2*Ntraj*sizeof(float));
	hipMalloc(&Stock, 2*Ntraj*sizeof(float));
	hipMalloc(&It, Ntraj*sizeof(float));


	PostInitDataCMRG();

	hipEventCreate(&start);			// GPU timer instructions
	hipEventCreate(&stop);				// GPU timer instructions
	hipEventRecord(start,0);			// GPU timer instructions

	// Step 3:
	//--------
	// Uncomment after memory allocation and free
	 MC_k<<<512,512>>>(Stock, x_0, r, vol, dt, 
					  P1, P2, K, res1, res2, Rando, 
					  It, B, Ntraj, M, CMRG);

	hipEventRecord(stop,0);			// GPU timer instructions
	hipEventSynchronize(stop);			// GPU timer instructions
	hipEventElapsedTime(&Tim,			// GPU timer instructions
			 start, stop);				// GPU timer instructions
	hipEventDestroy(start);			// GPU timer instructions
	hipEventDestroy(stop);				// GPU timer instructions

	hipMemcpy(res1C, res1, Ntraj*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(res2C, res2, Ntraj*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(res1);
	hipFree(res2);

   /************************************************************
	Step 2:
	-------
		Free appropriately Rando, Stock and It

   *************************************************************/

	hipFree(Rando);
	hipFree(Stock);
	hipFree(It);

	for(int i=0; i<Ntraj; i++){
		sum  += res1C[i]/Ntraj;
		sum2  += res2C[i]/Ntraj;
	}

	free(res1C);
	free(res2C);
	printf("The price is equal to %f\n", sum);

	printf("error associated to a confidence interval of 95%% = %f\n", 
		   1.96*sqrt((double)(1.0f/(Ntraj-1))*(Ntraj*sum2 - (sum*sum)))/sqrt((double)Ntraj));
	printf("Execution time %f ms\n", Tim);

	FreeCMRG();
	
	return 0;
}